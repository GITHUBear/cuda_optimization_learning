#include <hip/hip_runtime.h>
#include <iostream>
#include <atomic>

// Error checking macro and function
#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void addKernel(int *a, int *b, int *c, int N, int t) {
    if (t == 1) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < N) {
            c[idx] = a[idx] + b[idx];
        }
    } else {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < N) {
            c[idx] = a[idx] + b[idx];
        }
    }
}

__global__ void syncKernel(int *sync) {
    while (*((volatile int*)(sync)) != 1) {
        __threadfence_system();
    }
}

void logMemoryStatus(const char* message) {
    // Memory information variables
    size_t free_mem, total_mem;
    cudaCheck(hipMemGetInfo(&free_mem, &total_mem));
    float free_gb = free_mem / (float)(1 << 30);  // Convert bytes to gigabytes
    float total_gb = total_mem / (float)(1 << 30);

    // Variables for graph memory attributes
    size_t usedMemCurrent, usedMemHigh, reservedMemCurrent, reservedMemHigh;

    // Retrieve graph memory usage information
    cudaCheck(hipDeviceGetGraphMemAttribute(0, hipGraphMemAttrUsedMemCurrent, &usedMemCurrent));
    cudaCheck(hipDeviceGetGraphMemAttribute(0, hipGraphMemAttrUsedMemHigh, &usedMemHigh));
    cudaCheck(hipDeviceGetGraphMemAttribute(0, hipGraphMemAttrReservedMemCurrent, &reservedMemCurrent));
    cudaCheck(hipDeviceGetGraphMemAttribute(0, hipGraphMemAttrReservedMemHigh, &reservedMemHigh));

    // Print basic memory info
    std::cout << message << " - Free Memory: " << free_gb << " GB, Total Memory: " << total_gb << " GB, Graph Memory Usage: " << usedMemCurrent / (double)(1 << 30) << " GB, Graph Reserved Memory: " << reservedMemCurrent / (double)(1 << 30) << " GB\n";
}

struct HostFuncParams {
    int* c_ptr;
    int N;
    int* sync_data;
};

void host_func(HostFuncParams* params) {
    printf("============ host function called ============\n");
    for (int i = 0; i < params->N; ++i) {
        (params->c_ptr)[i] <<= 1;
    }
    asm volatile("mfence" ::: "memory");
    *((volatile int*)(params->sync_data)) = 1;
    printf("============ host function end ============\n");
}

int main() {
    hipMemPool_t mempool;
    hipDeviceGetDefaultMemPool(&mempool, 0);
    uint64_t threshold = 0; // UINT64_MAX;
    hipMemPoolSetAttribute(mempool, hipMemPoolAttrReleaseThreshold, &threshold);

    const int N = 1024 * 1024 * 256;
    const int bytes = N * sizeof(int);
    int *a, *b, *c, *h_c;

    int *sync_data;
    cudaCheck(hipHostAlloc(&sync_data, sizeof(int), hipHostMallocMapped));
    *sync_data = 0;

    int *d_sync_data;
    cudaCheck(hipHostGetDevicePointer(&d_sync_data, sync_data, 0));

    // Allocate device memory for a and b
    cudaCheck(hipMalloc(&a, bytes));
    cudaCheck(hipMalloc(&b, bytes));
    cudaCheck(hipMalloc(&c, bytes));

    // Initialize a and b on the host
    // int *h_a = new int[N];
    // int *h_b = new int[N];
    int *h_a, *h_b;
    hipHostMalloc(&h_a, N * sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&h_b, N * sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&h_c, N * sizeof(int), hipHostMallocDefault);
    for (int i = 0; i < N; ++i) {
        h_a[i] = i;
        h_b[i] = i;
    }

    // Copy data from host to device
    cudaCheck(hipMemcpy(a, h_a, bytes, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(b, h_b, bytes, hipMemcpyHostToDevice));

    // Allocate host memory for the result
    // h_c = new int[N];

    // Create a stream
    hipStream_t stream;
    cudaCheck(hipStreamCreate(&stream));

    logMemoryStatus("before capture");

    HostFuncParams param;
    param.c_ptr = h_c;
    param.N = N;
    param.sync_data = sync_data;

    hipEvent_t event;
    hipEventCreate(&event);

    // Begin graph capture
    hipGraph_t graph;
    cudaCheck(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));

    // Allocate memory for c during graph capture using cudaMallocAsync
    // cudaCheck(cudaMallocAsync(&c, bytes, stream));

    logMemoryStatus("inside capture, after hipMallocAsync");

    // Launch the add kernel
    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);
    addKernel<<<grid, block, 0, stream>>>(a, b, c, N, 1);

    // Copy the output to CPU using cudaMemcpyAsync
    cudaCheck(hipMemcpyAsync(h_c, c, bytes, hipMemcpyDeviceToHost, stream));

    // host_func(&param);
    hipLaunchHostFunc(stream, (hipHostFn_t)host_func, (void*)(&param));
    // cudaCheck(cudaStreamSynchronize(stream));

    // cudaMemPrefetchAsync()

    // syncKernel<<<1,1,0,stream>>>(d_sync_data);

    cudaCheck(hipMemcpyAsync(a, h_c, bytes, hipMemcpyHostToDevice, stream));

    hipEventRecord(event, stream);
    hipStreamWaitEvent(stream, event);

    addKernel<<<grid, block, 0, stream>>>(a, b, c, N, 2);

    cudaCheck(hipMemcpyAsync(h_c, c, bytes, hipMemcpyDeviceToHost, stream));

    // cudaCheck(cudaStreamSynchronize(stream));

    // End graph capture
    cudaCheck(hipStreamEndCapture(stream, &graph));

    // Launch the graph
    hipGraphExec_t graphExec;
    cudaCheck(hipGraphInstantiateWithFlags(&graphExec, graph));

    // logMemoryStatus("before execution");

    cudaCheck(hipGraphLaunch(graphExec, stream));
    cudaCheck(hipStreamSynchronize(stream));
    // logMemoryStatus("after the first execution");

    // Check result
    bool correct = true;
    for (int i = 0; i < N; ++i) {
        if (h_c[i] != 2*(h_a[i] + h_b[i]) + h_b[i]) {
            correct = false;
            printf("%d: %d expected, %d get\n", i, 2*(h_a[i] + h_b[i])+h_b[i], h_c[i]);
            break;
        }
    }
    if (correct) {
        std::cout << "Results are correct!" << std::endl;
    } else {
        std::cout << "Results are incorrect!" << std::endl;
    }

    // Cleanup

    cudaCheck(hipGraphDestroy(graph));
    cudaCheck(hipGraphExecDestroy(graphExec));
    cudaCheck(hipEventDestroy(event));

    cudaCheck(hipFree(a));
    cudaCheck(hipFree(b));
    cudaCheck(hipFree(c));
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    // delete[] h_a;
    // delete[] h_b;
    // delete[] h_c;

    cudaCheck(hipStreamDestroy(stream));

    return 0;
}