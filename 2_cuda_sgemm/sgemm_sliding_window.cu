#include "hip/hip_runtime.h"
#include "../utils/cuda_context.cuh"
#include <cstdio>

#define OFFSET(col, x, y) ((col) * (x) + (y))

void rand_matrix(int m, int n, float* M) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            M[OFFSET(n, i, j)] = 2.0 * (float)drand48() - 1.0;
        }
    }
}

void cpu_gemm(int M, int N, int K, float* A, float* B, float* C) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            C[OFFSET(N, i, j)] = 0.0;
            for (int k = 0; k < K; ++k) {
                C[OFFSET(N, i, j)] += A[OFFSET(K, i, k)] * B[OFFSET(N, k, j)];
            }
        }
    }
}

float compare_matrics(int M, int N, float *A, float *B) {
    float max_diff = 0.0;

    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            max_diff = std::max(max_diff, std::abs(A[OFFSET(N, i, j)] - B[OFFSET(N, i, j)]));
        }
    }
    return max_diff;
}

// A: M * K
// B: K * N
__global__ void sgemm(int M, int N, int K, float* A, float* B, float* C) {
    extern __shared__ float smem[];
    int block_size = blockDim.x;
    float* sub_A = smem;  // blockDim.x, blockDim.x
    float* sub_B = sub_A + block_size * block_size;  // blockDim.x, blockDim.x

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float tmp = 0.0;
    for (int s = 0; s < K; s += blockDim.x) {
        if (x < M && s + threadIdx.y < K)
            sub_A[OFFSET(block_size, threadIdx.x, threadIdx.y)] = A[OFFSET(K, x, s + threadIdx.y)];
        if (s + threadIdx.x < K && y < N)
            sub_B[OFFSET(block_size, threadIdx.x, threadIdx.y)] = B[OFFSET(N, s + threadIdx.x, y)];
        __syncthreads();

        if (x < M && y < N) {
            for (int k = 0; k < block_size && s + k < K; ++k) {
                tmp += sub_A[OFFSET(block_size, threadIdx.x, k)] * sub_B[OFFSET(block_size, k, threadIdx.y)];
            }
        }
        __syncthreads();
    }
    if (x < M && y < N) {
        C[OFFSET(N, x, y)] = tmp;
    }
}

void print_matrix(int M, int N, float* A) {
    printf("matrix:\n");
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f, ", A[OFFSET(N, i, j)]);
        }
        printf("\n");
    }
}

int main() {
    size_t M = 255;
    size_t N = 257;
    constexpr size_t K = 250;

    float* A, *B, *C, *CPU_C;
    hipMallocManaged((void**)&A, M * K * sizeof(float));
    hipMallocManaged((void**)&B, K * N * sizeof(float));
    hipMallocManaged((void**)&C, M * N * sizeof(float));
    CPU_C = (float*)malloc(M * N * sizeof(float));
    memset(C, 0, sizeof(float) * M * N);
    rand_matrix(M, K, A);
    rand_matrix(K, N, B);

    // print_matrix(M, K, A);
    // print_matrix(K, N, B);

    cpu_gemm(M, N, K, A, B, CPU_C);
    // print_matrix(M, N, CPU_C);

    constexpr size_t BLOCK_SIZE = 16;
    dim3 blk(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((M + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    constexpr size_t smem_size = 2 * BLOCK_SIZE * BLOCK_SIZE * sizeof(float);
    sgemm<<<grid, blk, smem_size>>>(M, N, K, A, B, C);
    hipDeviceSynchronize();
    // print_matrix(M, N, C);

    float max_diff = compare_matrics(M, N, CPU_C, C);
    if (max_diff > 0.5) {
        printf("result error!\n");
    } else {
        printf("check success!\n");
    }

    free(CPU_C);
    hipFree(A);
    hipFree(B);
    hipFree(C);
    return 0;
}